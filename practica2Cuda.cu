#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : practica2Cuda.cu
 Author      : Sergio Rosello & César Gil
 Version     : 0.1
 Copyright   : If you copy you will fail
 Description : Optimizaciones usando GPU
 ============================================================================
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "debug_time.h"
//#define LADO 100 //lado de la matriz

using namespace std;

// PARA LA PRACTICA DE CONCURRENCIA

/*
 *  1 - GUARDAR MATRIZ RESULTADO EN FICHERO
 *	2 -
 * */


// Matriz identidad
void inicializarMatrizIdentidad(float** matriz, int tamFilas, int tamColumnas){
	// filas y colunmnas deben ser iguales para una matriz identidad
	int i, j;
	for (i = 0; i < tamFilas; i++)
			for (j = 0; j < tamColumnas; j++)
				if(i==j){
					matriz[i][j] = 1;
				}else{
					matriz[i][j] = 0;
				}
}

// Matriz Aleatoria
void inicializarMatrizAleatoria(float **matriz, int tamFilas, int tamColumnas) {
	int i, j;

	for (i = 0; i < tamFilas; i++){
			for (j = 0; j < tamColumnas; j++){
				matriz[i][j] = rand() % 10;
				//matriz[i][j] = 1;
			}
	}
}


void crearMatriz(const char* nombre, bool identidad, int LADO)
{
	int i = 0;
	//int j = 0;
	//Inicializamos memoria para la matriz.
	float **mat = (float **)malloc(sizeof(float *) * LADO);
	for (i = 0; i < LADO; i++) {
		mat[i] = (float *)malloc(sizeof(float) * LADO);
	}

	//Dependiendo de si es identidad hace una llamada u otra
	if(identidad){
		inicializarMatrizIdentidad(mat, LADO, LADO);
	}else{
		inicializarMatrizAleatoria(mat, LADO, LADO);
	}

	//Abrimos el fichero binario mat en modo de escritura.
	FILE* fich_bin = fopen(nombre, "w");

	//Escribimos en el archivo binario apuntado por fich_bin

	//Volcamos los datos de la matriz en el archivo binario apuntado por fich_bin
	for (i = 0; i < LADO; i++)
		fwrite(mat[i], sizeof(int), LADO, fich_bin);
	fclose(fich_bin);

	//Liberamos memoria de cada uno de los elementos de la matriz.
	for (i = 0; i < LADO; i++) {
		free(mat[i]);
	}

	//Liberamos memoria de la matriz.
	free(mat);
}

//Guardar matriz resultado en fichero
void guardarMatrizResultado(int LADO, float** matriz){
	int i = 0;

	//Abrimos el fichero binario mat en modo de escritura.
	FILE* fich_bin = fopen("resultado.bin", "w");

	//Escribimos en el archivo binario apuntado por fich_bin
	//Volcamos los datos de la matriz en el archivo binario apuntado por fich_bin
	for (i = 0; i < LADO; i++){
		for (int j = 0; j < LADO; j++){
			fwrite(&(matriz[j][i]), sizeof(int), 1, fich_bin);
		}
	}
	fclose(fich_bin);

}


//Imprime la matriz recibida
void imprimirMatriz(float **matriz, int LADO) {
	for (int i = 0; i < LADO; i++) {
		for (int j = 0; j < LADO; j++)
			printf("%.0f ", matriz[i][j]);
		printf("\n");
	}
}

//lee la matriz de un fichero binario
void leerDatosBin(const char *nombreFichero, float ***datos, bool leerTraspuesta, int LADO) {
	FILE* fichero = fopen(nombreFichero, "r");
	//Funciones accesibles: fclose, fread, fwrite
	float **datosLeidos;
	//int numFilas, numColumnas;
	//sint i = 0, j = 0;


	//Inicializamos un array para guardar todos los datos que leemos del fichero.
	datosLeidos = (float **)malloc(LADO * sizeof(float*));

	//multiplicamos por 4 (bytes que ocupa un float)
	for (int i = 0; i < LADO; i++)
		datosLeidos[i] = (float*)malloc(sizeof(float)*LADO);

	if(!leerTraspuesta)
		for (int i = 0; i < LADO; i++)
			fread(datosLeidos[i], sizeof(float), LADO, fichero);

	else //Leer la matriz de una forma traspuesta.
		for (int i = 0; i < LADO; i++)
			for (int j = 0; j < LADO; j++)
				fread(&(datosLeidos[j][i]), sizeof(float), 1, fichero);


	//Derreferenciación.
	(*datos) = datosLeidos;
	fclose(fichero);
}

//lamada de la gpu para multiplicar los vectores
__device__ float multiplicarVectores(int lado, float* fila, float* columna){
	if(threadIdx.x == 0 && threadIdx.y == 0)
		printf("Multiplicando vectores\n");
	float resultadoAux = 0;
	for(int i = 0; i<lado; i++){
		resultadoAux += fila[i] * columna[i];
	}

	return resultadoAux;

}

//llamada a la gpu para multiplicar matrices
__global__ void kernel_multiplicarMatrices(int lado, float** matriz1, float** matriz2, float** resultado){
	//printf("estoy multiplicando\n");
	int fila = blockIdx.x * blockDim.y + threadIdx.y;
	int columna = blockIdx.y * blockDim.x + threadIdx.x;

	//control de errores del thread
	if((fila >= lado) || (columna >= lado)){
		//printf("ha ocurrido un error en multiplicacion\n");
		return;
		}

	resultado[fila][columna] = multiplicarVectores(lado, matriz1[fila], matriz2[columna]);
	printf("cuda thread %d %d %.0f \n",fila,columna,resultado[fila][columna]);
}



int main(int argc, char** argv){
	// inicializacion del debug_time
	DEBUG_TIME_INIT;
	DEBUG_TIME_START;

	//Se asigna el lado de la matriz segun el parametro introducido en la ejecucion
	int LADO = atoi(argv[1]);

	//Inicialización de la semilla para los números aleatorios.
	srand(time(NULL));

	bool leerTraspuesta = true;

	//CREACION DE LAS MATRICES ALEATORIAS EN UN FICHERO BINARIO
	crearMatriz("mat.bin", false, LADO);
	crearMatriz("matIdentidad.bin", true, LADO);

	//CARGA E INICIALIZACION DE LAS MATRICES
	//CPU
	float** matriz1_host;
	float** matriz2_host;
	float** matrizResultado_host;

	//NEXO (memoria intermedia)
	float** matriz1_nexo;
	float** matriz2_nexo;
	float** matrizResultado_nexo;

	//GPU
	float** matriz1_device;
	float** matriz2_device;
	float** matrizResultado_device;

	//leemos de fichero binario
	leerDatosBin("mat.bin", &matriz1_host, leerTraspuesta, LADO);
	leerDatosBin("matIdentidad.bin", &matriz2_host, !leerTraspuesta, LADO);

	//IMPRIME LAS MATRICES GENERADAS
	printf("Se van a generar matrices de %d X %d : \n", LADO, LADO);

	printf("MATRIZ A: \n\n");
	imprimirMatriz(matriz1_host, LADO);

	printf("MATRIZ B Identidad: \n\n");
	imprimirMatriz(matriz2_host, LADO);

	//Reserva para el resultado del host
	matrizResultado_host = (float**)malloc(LADO * sizeof(float*));
	for(int i=0; i < LADO; i++){
		matrizResultado_host[i] = (float*)malloc(LADO * sizeof(float));
	}

	//Reserva de la memoria intermedia
	matriz1_nexo = (float**)malloc(LADO * sizeof(float*));
	matriz2_nexo = (float**)malloc(LADO * sizeof(float*));
	matrizResultado_nexo = (float**)malloc(LADO * sizeof(float*));

	//Reserva de memoria en GPU
	hipError_t err1 = hipMalloc((void**)&matriz1_device, sizeof(float*)* LADO);
	printf("Run Kernel: %s \n", hipGetErrorString(err1));

	err1 = hipMalloc((void**)&matriz2_device, sizeof(float*)* LADO);
	printf("Run Kernel: %s \n", hipGetErrorString(err1));

	err1 = hipMalloc((void**)&matrizResultado_device, sizeof(float*)* LADO);
	printf("Run Kernel: %s \n", hipGetErrorString(err1));

	//Reserva de memoria para cada uno de los arrays intermedios
	for(int i = 0; i < LADO; i++){
		err1 = hipMalloc((void**)&matriz1_nexo[i], sizeof(float)* LADO);
		printf("matriz1_nexo Run Kernel: %s \n", hipGetErrorString(err1));

		err1 = hipMalloc((void**)&matriz2_nexo[i], sizeof(float)* LADO);
		printf("matriz2_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		hipMalloc((void**)&(matrizResultado_nexo[i]), sizeof(float)* LADO);
	}

	//Copia el contenido de los arrays de CPU a los arrays de la matriz intermedia
	for(int i = 0; i < LADO; i++){
			err1 = hipMemcpy(matriz1_nexo[i], matriz1_host[i], LADO * sizeof(float),hipMemcpyHostToDevice);
			printf("cudaMemcoy matriz2_host1 a nexo1 Run Kernel: %s \n", hipGetErrorString(err1));
			err1 = hipMemcpy(matriz2_nexo[i], matriz2_host[i], LADO * sizeof(float),hipMemcpyHostToDevice);
			printf("cudaMemcoy matriz2_host2 a nexo2 Run Kernel: %s \n", hipGetErrorString(err1));
	}

	//copia el contenido del array de punteros de CPU a GPU
	err1 = hipMemcpy(matriz1_device, matriz1_nexo, LADO * sizeof(float*),hipMemcpyHostToDevice);
	printf("copia de cpu a gpu array de punteros matriz1 Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMemcpy(matriz2_device, matriz2_nexo, LADO * sizeof(float*),hipMemcpyHostToDevice);
	printf("copia de cpu a gpu array de punteros matriz2 Run Kernel: %s \n", hipGetErrorString(err1));
	hipMemcpy(matrizResultado_device, matrizResultado_nexo, LADO * sizeof(float*),hipMemcpyHostToDevice);

	//Operaciones en GPU:
	// tamBloque = 32 porque los kernels proporcionan las instrucciones en warps (32 threads)
	//entonces tiene que ser multiplo de 32 para no despediciar threads.
	// dimensionGrid -> (LADO / tamBloque) + 1 para calcular el numero de bloques para la x y para y,  1 para la z
	// dimensionBlock -> numero de threads por cada bloque (32*32 = 1024 threads)
	// https://codeyarns.com/2011/02/16/cuda-dim3/
	// http://www.icl.utk.edu/~mgates3/docs/cuda.html
	int tamBloque = 32;

	dim3 dimensionGrid = dim3((int)(LADO / tamBloque) + 1, (int)(LADO / tamBloque) + 1, 1);
	dim3 dimensionBlock = dim3(tamBloque, tamBloque, 1);

	printf("Antes de multiplicar\n");
	//hace la multiplicacion en GPU
	{
		// PARA EL CALCULO DEL TIEMPO DE EJECUCION EN GPU
		DEBUG_TIME_INIT;
		DEBUG_TIME_START;

		kernel_multiplicarMatrices <<<dimensionGrid,dimensionBlock>>>(LADO, matriz1_device, matriz2_device, matrizResultado_device);

	//Para que espere hasta que todos los threads terminen (CUDA THREADS SYNCRONIZE)
		hipError_t error = hipDeviceSynchronize();
		printf("Thread synchronization: %s \n", hipGetErrorString(error));

		//Finaliza el tiempo de ejecucion en GPU
		DEBUG_TIME_END;

		// Imprimir el tiempo
		DEBUG_PRINT_FINALTIME("Tiempo transcurrido en GPU de multiplicar matrices: \n\t");
	}
	//pasamos el resultado de device al host
	for(int i = 0; i < LADO; i++){
		err1 = hipMemcpy(matrizResultado_host[i], matrizResultado_nexo[i], LADO * sizeof(float),hipMemcpyDeviceToHost);
		printf("copia de gpu a cpu final Run Kernel: %s \n", hipGetErrorString(err1));
	}

	//imprime la matriz resultado una vez copiada al host
	printf("El resultado es: \n");
	imprimirMatriz(matrizResultado_host, LADO);

	//Guarda el resultado en un fichero binario
	guardarMatrizResultado(LADO, matrizResultado_host);

	/*
	// **************   TEST PARA PROBAR QUE SE HA GUADADO BIEN ******
	float** test;
	test = (float**)malloc(LADO * sizeof(float*));
		for(int i=0; i < LADO; i++){
			test[i] = (float*)malloc(LADO * sizeof(float));
	}

	leerDatosBin("resultado.bin", &test, leerTraspuesta, LADO);
	printf("\n RESULTADO DE RESUTLADO\n");
	imprimirMatriz(test, LADO);*/

	//LIBERACION DE MEMORIA DE CPU E INTERMEDIA
	for(int i = 0; i < LADO; i++){
		//CPU
		free(matriz1_host[i]);
		free(matriz2_host[i]);
		free(matrizResultado_host[i]);

		//Intermedia
		hipFree(matriz1_nexo[i]);
		hipFree(matriz2_nexo[i]);
		hipFree(matrizResultado_nexo[i]);
	}

	//liberacion del array de punteros
	free(matriz1_host);
	free(matriz2_host);
	free(matrizResultado_host);

	free(matriz1_nexo);
	free(matriz2_nexo);
	free(matrizResultado_nexo);

	//free GPU
	hipFree(matriz1_device);
	hipFree(matriz2_device);
	hipFree(matrizResultado_device);


	DEBUG_TIME_END;
	DEBUG_PRINT_FINALTIME("Tiempo total del programa: \n\t");
}

